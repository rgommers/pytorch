#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

namespace at {
namespace native {
namespace {

template <typename scalar_t, typename accscalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void upsample_linear1d_out_frame(
    const int64_t n,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor<scalar_t, 3> idata,
    PackedTensorAccessor<scalar_t, 3> odata) {
  int64_t index = threadIdx.x + blockIdx.x * blockDim.x;

  const int64_t batchsize = idata.size(0);
  const int64_t channels = idata.size(1);
  const int64_t width1 = idata.size(2);
  const int64_t width2 = odata.size(2);

  if (index < n) {
    const int64_t w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int64_t w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][w1];
          odata[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int64_t w1 = w1r;
    const int64_t w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val =
            w0lambda * idata[n][c][w1] + w1lambda * idata[n][c][w1 + w1p];
        odata[n][c][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void upsample_linear1d_out_frame_backward(
    const int64_t n,
    const accscalar_t rwidth,
    const bool align_corners,
    PackedTensorAccessor<scalar_t, 3> idata,
    const PackedTensorAccessor<scalar_t, 3> odata) {
  int64_t index = threadIdx.x + blockIdx.x * blockDim.x;

  const int64_t batchsize = idata.size(0);
  const int64_t channels = idata.size(1);
  const int64_t width1 = idata.size(2);
  const int64_t width2 = odata.size(2);

  if (index < n) {
    const int64_t w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int64_t w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = odata[n][c][w1];
          idata[n][c][w2] += val;
        }
      }
      return;
    }
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int64_t w1 = w1r;
    const int64_t w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t d2val = odata[n][c][w2];
        atomicAdd(&idata[n][c][w1], static_cast<scalar_t>(w0lambda * d2val));
        atomicAdd(
            &idata[n][c][w1 + w1p], static_cast<scalar_t>(w1lambda * d2val));
      }
    }
  }
}

static void upsample_linear1d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners) {
  checkAllSameGPU("upsample_linear1d_out_cuda", {input, output});

  AT_CHECK(
      output_size.size() == 1,
      "It is expected output_size equals to 1, but got size ",
      output_size.size());

  int64_t output_width = output_size[0];

  int64_t nbatch = input.size(0);
  int64_t channels = input.size(1);
  int64_t input_width = input.size(2);

  upsample_1d_shape_check(
      input, Tensor(), nbatch, channels, input_width, output_width);

  output.resize_({input.size(0), input.size(1), output_width});
  output.zero_();

  auto idata = input.packed_accessor<scalar_t, 3>();
  auto odata = output.packed_accessor<scalar_t, 3>();

  AT_ASSERT(input_width > 0 && output_width > 0);

  const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
      input_width, output_width, align_corners);
  const int64_t num_kernels = output_width;
  const int64_t num_threads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_linear1d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        upsample_linear1d_out_frame<scalar_t, accscalar_t>
            <<<(num_kernels + num_threads - 1) / num_threads,
               num_threads,
               0,
               stream>>>(num_kernels, rwidth, align_corners, idata, odata);
      });

  AT_CHECK(
      hipGetLastError() == hipSuccess,
      "Failed with error code ",
      hipGetLastError());
}

static void upsample_linear1d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners) {
  checkAllSameGPU(
      "upsample_linear1d_backward_out_cuda", {grad_output_, grad_input});

  AT_CHECK(
      output_size.size() == 1,
      "It is expected output_size equals to 1, but got size ",
      output_size.size());

  AT_CHECK(
      input_size.size() == 3,
      "It is expected input_size equals to 3, but got size ",
      input_size.size());

  int64_t output_width = output_size[0];

  int64_t nbatch = input_size[0];
  int64_t channels = input_size[1];
  int64_t input_width = input_size[2];

  upsample_1d_shape_check(
      Tensor(), grad_output_, nbatch, channels, input_width, output_width);

  Tensor grad_output = grad_output_.contiguous();

  grad_input.resize_({nbatch, channels, input_width});
  grad_input.zero_();

  auto idata = grad_input.packed_accessor<scalar_t, 3>();
  auto odata = grad_output.packed_accessor<scalar_t, 3>();

  const int64_t num_kernels = output_width;
  const int64_t num_threads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "upsample_linear1d_out_frame_backward", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners);

        upsample_linear1d_out_frame_backward<scalar_t, accscalar_t>
            <<<(num_kernels + num_threads - 1) / num_threads,
               num_threads,
               0,
               stream>>>(num_kernels, rwidth, align_corners, idata, odata);
      });

  AT_CHECK(
      hipGetLastError() == hipSuccess,
      "Failed with error code ",
      hipGetLastError());
}

} // namespace

Tensor& upsample_linear1d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners) {
  upsample_linear1d_out_cuda_template(
      output, input, output_size, align_corners);
  return output;
}

Tensor upsample_linear1d_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners) {
  Tensor output = at::empty({0}, input.options());
  upsample_linear1d_out_cuda_template(
      output, input, output_size, align_corners);
  return output;
}

Tensor& upsample_linear1d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners) {
  upsample_linear1d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners);
  return grand_input;
}

Tensor upsample_linear1d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners) {
  Tensor grad_input = at::zeros(input_size, grad_output.options());
  upsample_linear1d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners);
  return grad_input;
}

} // namespace native
} // namespace at
