#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

namespace at {
namespace native {
namespace {

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_trilinear3d_out_frame(
    const int64_t n,
    const accscalar_t rdepth,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor<scalar_t, 5> idata,
    PackedTensorAccessor<scalar_t, 5> odata) {
  int64_t index = threadIdx.x + blockIdx.x * blockDim.x;

  const int64_t batchsize = idata.size(0);
  const int64_t channels = idata.size(1);
  const int64_t depth1 = idata.size(2);
  const int64_t height1 = idata.size(3);
  const int64_t width1 = idata.size(4);
  const int64_t depth2 = odata.size(2);
  const int64_t height2 = odata.size(3);
  const int64_t width2 = odata.size(4);

  if (index < n) {
    const int64_t w2 = (index % (height2 * width2)) % width2; // 0:width2-1
    const int64_t h2 = (index % (height2 * width2)) / width2; // 0:height2-1
    const int64_t t2 = index / (height2 * width2); // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int64_t t1 = t2;
      const int64_t h1 = h2;
      const int64_t w1 = w2;

      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][t1][h1][w1];
          odata[n][c][t2][h2][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t t1r = linear_upsampling_compute_source_index<accscalar_t>(
        rdepth, t2, align_corners);
    const int64_t t1 = t1r;
    const int64_t t1p = (t1 < depth1 - 1) ? 1 : 0;
    const accscalar_t t1lambda = t1r - t1;
    const accscalar_t t0lambda = static_cast<accscalar_t>(1) - t1lambda;
    //
    const accscalar_t h1r = linear_upsampling_compute_source_index<accscalar_t>(
        rheight, h2, align_corners);
    const int64_t h1 = h1r;
    const int64_t h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = linear_upsampling_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners);
    const int64_t w1 = w1r;
    const int64_t w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val = t0lambda *
                (h0lambda *
                     (w0lambda * idata[n][c][t1][h1][w1] +
                      w1lambda * idata[n][c][t1][h1][w1 + w1p]) +
                 h1lambda *
                     (w0lambda * idata[n][c][t1][h1 + h1p][w1] +
                      w1lambda * idata[n][c][t1][h1 + h1p][w1 + w1p])) +
            t1lambda *
                (h0lambda *
                     (w0lambda * idata[n][c][t1 + t1p][h1][w1] +
                      w1lambda * idata[n][c][t1 + t1p][h1][w1 + w1p]) +
                 h1lambda *
                     (w0lambda * idata[n][c][t1 + t1p][h1 + h1p][w1] +
                      w1lambda * idata[n][c][t1 + t1p][h1 + h1p][w1 + w1p]));
        odata[n][c][t2][h2][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_trilinear3d_backward_out_frame(
    const int64_t n,
    const accscalar_t rdepth,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    PackedTensorAccessor<scalar_t, 5> idata,
    const PackedTensorAccessor<scalar_t, 5> odata) {
  int64_t index = threadIdx.x + blockIdx.x * blockDim.x;

  const int64_t batchsize = idata.size(0);
  const int64_t channels = idata.size(1);
  const int64_t depth1 = idata.size(2);
  const int64_t height1 = idata.size(3);
  const int64_t width1 = idata.size(4);
  const int64_t depth2 = odata.size(2);
  const int64_t height2 = odata.size(3);
  const int64_t width2 = odata.size(4);

  if (index < n) {
    const int64_t w2 = (index % (height2 * width2)) % width2; // 0:width2-1
    const int64_t h2 = (index % (height2 * width2)) / width2; // 0:height2-1
    const int64_t t2 = index / (height2 * width2); // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int64_t t1 = t2;
      const int64_t h1 = h2;
      const int64_t w1 = w2;

      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = odata[n][c][t1][h1][w1];
          idata[n][c][t2][h2][w2] += val;
        }
      }
      return;
    }
    //
    const accscalar_t t1r = linear_upsampling_compute_source_index<accscalar_t>(
        rdepth, t2, align_corners);
    const int64_t t1 = t1r;
    const int64_t t1p = (t1 < depth1 - 1) ? 1 : 0;
    const accscalar_t t1lambda = t1r - t1;
    const accscalar_t t0lambda = static_cast<accscalar_t>(1) - t1lambda;
    //
    const accscalar_t h1r = linear_upsampling_compute_source_index<accscalar_t>(
        rheight, h2, align_corners);
    const int64_t h1 = h1r;
    const int64_t h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = linear_upsampling_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners);
    const int64_t w1 = w1r;
    const int64_t w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t d2val = odata[n][c][t2][h2][w2];
        atomicAdd(
            &idata[n][c][t1][h1][w1],
            static_cast<scalar_t>(t0lambda * h0lambda * w0lambda * d2val));
        atomicAdd(
            &idata[n][c][t1][h1][w1 + w1p],
            static_cast<scalar_t>(t0lambda * h0lambda * w1lambda * d2val));
        atomicAdd(
            &idata[n][c][t1][h1 + h1p][w1],
            static_cast<scalar_t>(t0lambda * h1lambda * w0lambda * d2val));
        atomicAdd(
            &idata[n][c][t1][h1 + h1p][w1 + w1p],
            static_cast<scalar_t>(t0lambda * h1lambda * w1lambda * d2val));
        atomicAdd(
            &idata[n][c][t1 + t1p][h1][w1],
            static_cast<scalar_t>(t1lambda * h0lambda * w0lambda * d2val));
        atomicAdd(
            &idata[n][c][t1 + t1p][h1][w1 + w1p],
            static_cast<scalar_t>(t1lambda * h0lambda * w1lambda * d2val));
        atomicAdd(
            &idata[n][c][t1 + t1p][h1 + h1p][w1],
            static_cast<scalar_t>(t1lambda * h1lambda * w0lambda * d2val));
        atomicAdd(
            &idata[n][c][t1 + t1p][h1 + h1p][w1 + w1p],
            static_cast<scalar_t>(t1lambda * h1lambda * w1lambda * d2val));
      }
    }
  }
}

static void upsample_trilinear3d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};

  checkAllSameGPU("upsample_trilinear3d_out_cuda", {input_arg, output_arg});

  AT_CHECK(
      output_size.size() == 3,
      "It is expected output_size equals to 3, but got size ",
      output_size.size());

  int64_t output_depth = output_size[0];
  int64_t output_height = output_size[1];
  int64_t output_width = output_size[2];

  int64_t nbatch = input.size(0);
  int64_t channels = input.size(1);
  int64_t input_depth = input.size(2);
  int64_t input_height = input.size(3);
  int64_t input_width = input.size(4);

  upsample_3d_shape_check(
      input,
      Tensor(),
      nbatch,
      channels,
      input_depth,
      input_height,
      input_width,
      output_depth,
      output_height,
      output_width);

  output.resize_({input.size(0),
                  input.size(1),
                  output_depth,
                  output_height,
                  output_width});
  output.zero_();

  AT_ASSERT(
      input_depth > 0 && input_height > 0 && input_width > 0 &&
      output_depth > 0 && output_height > 0 && output_width > 0);

  const int64_t num_kernels = output_depth * output_height * output_width;
  const int64_t num_threads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_trilinear3d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor<scalar_t, 5>();
        auto odata = output.packed_accessor<scalar_t, 5>();

        const accscalar_t rdepth = linear_upsampling_compute_scale<accscalar_t>(
            input_depth, output_depth, align_corners);
        const accscalar_t rheight =
            linear_upsampling_compute_scale<accscalar_t>(
                input_height, output_height, align_corners);
        const accscalar_t rwidth = linear_upsampling_compute_scale<accscalar_t>(
            input_width, output_width, align_corners);

        upsample_trilinear3d_out_frame<scalar_t, accscalar_t>
            <<<(num_kernels + num_threads - 1) / num_threads,
               num_threads,
               0,
               stream>>>(
                num_kernels,
                rdepth,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
      });

  AT_CHECK(
      hipGetLastError() == hipSuccess,
      "Failed with error code ",
      hipGetLastError());
}

static void upsample_trilinear3d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};

  checkAllSameGPU(
      "upsample_trilinear3d_backward_out_cuda",
      {grad_output_arg, grad_input_arg});

  AT_CHECK(
      output_size.size() == 3,
      "It is expected output_size equals to 3, but got size ",
      output_size.size());

  AT_CHECK(
      input_size.size() == 5,
      "It is expected input_size equals to 5, but got size ",
      input_size.size());

  int64_t output_depth = output_size[0];
  int64_t output_height = output_size[1];
  int64_t output_width = output_size[2];

  int64_t nbatch = input_size[0];
  int64_t channels = input_size[1];
  int64_t input_depth = input_size[2];
  int64_t input_height = input_size[3];
  int64_t input_width = input_size[4];

  upsample_3d_shape_check(
      Tensor(),
      grad_output_,
      nbatch,
      channels,
      input_depth,
      input_height,
      input_width,
      output_depth,
      output_height,
      output_width);
  Tensor grad_output = grad_output_.contiguous();

  grad_input.resize_(
      {nbatch, channels, input_depth, input_height, input_width});
  grad_input.zero_();

  const int64_t num_kernels = output_depth * output_height * output_width;
  const int64_t num_threads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(),
      "upsample_trilinear3d_backward_out_frame",
      [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.packed_accessor<scalar_t, 5>();
        auto odata = grad_output.packed_accessor<scalar_t, 5>();

        const accscalar_t rdepth = linear_upsampling_compute_scale<accscalar_t>(
            input_depth, output_depth, align_corners);
        const accscalar_t rheight =
            linear_upsampling_compute_scale<accscalar_t>(
                input_height, output_height, align_corners);
        const accscalar_t rwidth = linear_upsampling_compute_scale<accscalar_t>(
            input_width, output_width, align_corners);

        upsample_trilinear3d_backward_out_frame<scalar_t, accscalar_t>
            <<<(num_kernels + num_threads - 1) / num_threads,
               num_threads,
               0,
               stream>>>(
                num_kernels,
                rdepth,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
      });

  AT_CHECK(
      hipGetLastError() == hipSuccess,
      "Failed with error code ",
      hipGetLastError());
}

} // namespace

Tensor& upsample_trilinear3d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners) {
  upsample_trilinear3d_out_cuda_template(
      output, input, output_size, align_corners);
  return output;
}

Tensor upsample_trilinear3d_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners) {
  Tensor output = at::empty({0}, input.options());
  upsample_trilinear3d_out_cuda_template(
      output, input, output_size, align_corners);
  return output;
}

Tensor& upsample_trilinear3d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners) {
  upsample_trilinear3d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners);
  return grad_input;
}

Tensor upsample_trilinear3d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners) {
  Tensor grad_input = at::zeros(input_size, grad_output.options());
  upsample_trilinear3d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners);
  return grad_input;
}

} // namespace native
} // namespace at
